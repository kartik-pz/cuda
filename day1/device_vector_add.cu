#include <stdio.h>
#include <hip/hip_runtime.h>

// Compute vector sum h_C = h_A + h_B

__global__ void VectorAddKernel(float* A, float* B, float* C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

int main() {
  const int n = 1024;
  const int size = n * sizeof(int);

  // Allocate host memory
  float *h_A, *h_B, *h_C;
  h_A = (float*) malloc(size);
  h_B = (float*) malloc(size);
  h_C = (float*) malloc(size);

  // Initialize host arrays
  for (int i = 0; i < n; ++i) {
    h_A[i] = 1000.0f + i;
    h_B[i] = 2000.0f - i;
  }

  // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

  // Copy data to the device
    hipMemcpy(d_A, h_A, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * sizeof(float), hipMemcpyHostToDevice);

  // Compute on the device
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    VectorAddKernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n);
    

  // Copy result back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    


  // Verify the result
  for (int i = 0; i < n; ++i) {
    if (h_C[i] != 3000.0f) {
      printf("Mismatch at element %d\n", i);
      return 1;
    }
  }

  printf("Success!\n");

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
  
  return 0;
}